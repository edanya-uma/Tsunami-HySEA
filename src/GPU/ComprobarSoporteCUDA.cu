
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" int comprobarSoporteCUDA()
{
    int valor, dev, deviceCount;

    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
        valor = 1;
	else {
		for (dev=0; dev < deviceCount; ++dev) {
	        hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, dev);
			if (deviceProp.major >= 1)
	            break;
	    }
		if (dev == deviceCount)
			valor = 2;
		else
			valor = 0;
	}
	return valor;
}
