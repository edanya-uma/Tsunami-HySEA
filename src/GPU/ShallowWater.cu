#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <cstring>
#include <fstream>
#include "helper_timer.h"
#include "Reduccion_kernel.cu"
#include "Volumen_kernel.cu"
#include "Arista_kernel.cu"
#include "Deformacion.cu"
#include "netcdf.cu"

/*****************/
/* NetCDF saving */
/*****************/

void guardarNetCDFNivel0(double2 *datosVolumenes_1, double2 *datosVolumenes_2, float *vec, int num, int num_volx,
				int num_voly, double Hmin, double tiempo_act, double epsilon_h, double H, double Q, double T)
{
	double tiempo = tiempo_act*T;
	double2 datos;
	double h;
	double maxheps, factor;
	int i;
	int num_volumenes = num_volx*num_voly;

	escribirTiempoNC(num, tiempo);
	for (i=0; i<num_volumenes; i++) {
		datos = datosVolumenes_1[i];
		vec[i] = (float) ( (datos.x < epsilon_h) ? -9999.0 : (datos.x - datos.y - Hmin)*H );
	}
	escribirEtaNC(num_volx, num_voly, num, vec);
	for (i=0; i<num_volumenes; i++) {
		h = datosVolumenes_1[i].x;
		if (h < epsilon_h) {
			maxheps = epsilon_h;
			factor = M_SQRT2*h / sqrt(h*h*h*h + maxheps*maxheps*maxheps*maxheps)*(Q/H);
		}
		else {
			factor = Q/(h*H);
		}
		vec[i] = (float) (datosVolumenes_2[i].x*factor);
	}
	escribirUxNC(num_volx, num_voly, num, vec);
	for (i=0; i<num_volumenes; i++) {
		h = datosVolumenes_1[i].x;
		if (h < epsilon_h) {
			maxheps = epsilon_h;
			factor = M_SQRT2*h / sqrt(h*h*h*h + maxheps*maxheps*maxheps*maxheps)*(Q/H);
		}
		else {
			factor = Q/(h*H);
		}
		vec[i] = (float) (datosVolumenes_2[i].y*factor);
	}
	escribirUyNC(num_volx, num_voly, num, vec);
}

/***************/
/* Time series */
/***************/

// Format of the NetCDF time series file:
// For each point:
//   <longitude>
//   <latitude>
//   <bathymetry (original if okada_flag is INITIAL_FROM_FILE; deformed if okada_flag is OKADA_STANDARD)
//   <minimum eta>
//   <maximum eta>
// For each time:
//   For each point:
//     <eta point 1> <u point 1> <v point 1> ... <eta point n> <u point n> <v point n>
void obtenerBatimetriaParaSerieTiempos(double2 *datosVolumenesNivel_1, int numPuntosGuardar,
		int *posicionesVolumenesGuardado, float *profPuntosGuardado, double Hmin, double H)
{
	int i;

	for (i=0; i<numPuntosGuardar; i++) {
		if (posicionesVolumenesGuardado[i] != -1)
			profPuntosGuardado[i] = (float) ((datosVolumenesNivel_1[i].y + Hmin)*H);
	}
}

void guardarSerieTiemposNivel0(double2 *datosVolumenesNivel_1, double2 *datosVolumenesNivel_2,
			int numPuntosGuardar, int *posicionesVolumenesGuardado, float *etaPuntosGuardado,
			float *uPuntosGuardado, float *vPuntosGuardado, float *etaMinPuntosGuardado,
			float *etaMaxPuntosGuardado, double Hmin, int num_ts, double tiempo_act,
			double H, double Q, double T)
{
	double tiempo = tiempo_act*T;
	double h;
	int i;

	for (i=0; i<numPuntosGuardar; i++) {
		if (posicionesVolumenesGuardado[i] != -1) {
			h = datosVolumenesNivel_1[i].x;
			etaPuntosGuardado[i] = (float) ((h - datosVolumenesNivel_1[i].y - Hmin)*H);
			uPuntosGuardado[i] = (float) (datosVolumenesNivel_2[i].x*Q/H);
			vPuntosGuardado[i] = (float) (datosVolumenesNivel_2[i].y*Q/H);
			etaMinPuntosGuardado[i] = min(etaPuntosGuardado[i], etaMinPuntosGuardado[i]);
			etaMaxPuntosGuardado[i] = max(etaPuntosGuardado[i], etaMaxPuntosGuardado[i]);
		}
	}
	writeStateTimeSeriesNC(num_ts, tiempo, numPuntosGuardar, etaPuntosGuardado, uPuntosGuardado, vPuntosGuardado);
}

/***************/
/* Blocks size */
/***************/

void obtenerTamBloquesKernel(int num_volx, int num_voly, dim3 *blockGridVer1, dim3 *blockGridVer2,
							dim3 *blockGridHor1, dim3 *blockGridHor2, dim3 *blockGridEst)
{
	int num_aristas_ver1, num_aristas_ver2;
	int num_aristas_hor1, num_aristas_hor2;

	num_aristas_ver1 = (num_volx/2 + 1)*num_voly;
	num_aristas_ver2 = ((num_volx&1) == 0) ? num_volx*num_voly/2 : num_aristas_ver1;
	num_aristas_hor1 = (num_voly/2 + 1)*num_volx;
	num_aristas_hor2 = ((num_voly&1) == 0) ? num_volx*num_voly/2 : num_aristas_hor1;

	blockGridVer1->x = iDivUp(num_aristas_ver1/num_voly, NUM_HEBRASX_ARI);
	blockGridVer1->y = iDivUp(num_voly, NUM_HEBRASY_ARI);
	blockGridVer2->x = iDivUp(num_aristas_ver2/num_voly, NUM_HEBRASX_ARI);
	blockGridVer2->y = iDivUp(num_voly, NUM_HEBRASY_ARI);

	blockGridHor1->x = iDivUp(num_volx, NUM_HEBRASX_ARI);
	blockGridHor1->y = iDivUp(num_aristas_hor1/num_volx, NUM_HEBRASY_ARI);
	blockGridHor2->x = iDivUp(num_volx, NUM_HEBRASX_ARI);
	blockGridHor2->y = iDivUp(num_aristas_hor2/num_volx, NUM_HEBRASY_ARI);

	blockGridEst->x = iDivUp(num_volx, NUM_HEBRASX_EST);
	blockGridEst->y = iDivUp(num_voly, NUM_HEBRASY_EST);
}

/*******************/
/* Free GPU memory */
/*******************/

void liberarMemoria(int numNiveles, double2 *d_datosVolumenesNivel_1, double2 *d_datosVolumenesNivel_2,
			tipoDatosSubmalla d_datosNivel, double *d_eta1MaximaNivel, double *d_deltaTVolumenesNivel,
			double2 *d_acumuladorNivel1_1, double2 *d_acumuladorNivel1_2, int leer_fichero_puntos,
			int *d_posicionesVolumenesGuardado, double2 *d_datosVolumenesGuardado_1, double2 *d_datosVolumenesGuardado_2)
{
	hipFree(d_datosNivel.areaYCosPhi);
	hipFree(d_datosNivel.anchoVolumenes);
	hipFree(d_datosNivel.altoVolumenes);
	hipFree(d_datosVolumenesNivel_1);
	hipFree(d_datosVolumenesNivel_2);
	hipFree(d_eta1MaximaNivel);
	hipFree(d_deltaTVolumenesNivel);
	hipFree(d_acumuladorNivel1_1);
	hipFree(d_acumuladorNivel1_2);
	if (leer_fichero_puntos == 1) {
		hipFree(d_posicionesVolumenesGuardado);
		hipFree(d_datosVolumenesGuardado_1);
		hipFree(d_datosVolumenesGuardado_2);
	}
}

/******************/
/* Main functions */
/******************/

double obtenerDeltaTInicialNivel0(double2 *d_datosVolumenesNivel0_1, double2 *d_datosVolumenesNivel0_2, tipoDatosSubmalla *d_datosNivel0,
		double *d_deltaTVolumenesNivel0, double2 *d_acumulador_1, int numVolxNivel0, int numVolyNivel0, double borde_izq,
		double borde_der, double borde_sup, double borde_inf, double CFL, double epsilon_h, dim3 blockGridVer1, dim3 blockGridVer2,
		dim3 blockGridHor1, dim3 blockGridHor2, dim3 threadBlockAri, dim3 blockGridEst, dim3 threadBlockEst)
{
	double delta_T;

	procesarAristasVerDeltaTInicialNivel0GPU<<<blockGridVer1, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, numVolxNivel0, numVolyNivel0, borde_izq, borde_der, d_datosNivel0->altoVolumenes,
		d_acumulador_1, epsilon_h, 1);
	procesarAristasVerDeltaTInicialNivel0GPU<<<blockGridVer2, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, numVolxNivel0, numVolyNivel0, borde_izq, borde_der, d_datosNivel0->altoVolumenes,
		d_acumulador_1, epsilon_h, 2);
	procesarAristasHorDeltaTInicialNivel0GPU<<<blockGridHor1, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		numVolxNivel0, numVolyNivel0, borde_sup, borde_inf, d_datosNivel0->anchoVolumenes, d_acumulador_1, epsilon_h, 1);
	procesarAristasHorDeltaTInicialNivel0GPU<<<blockGridHor2, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		numVolxNivel0, numVolyNivel0, borde_sup, borde_inf, d_datosNivel0->anchoVolumenes, d_acumulador_1, epsilon_h, 2);
	obtenerDeltaTVolumenesGPU<<<blockGridEst, threadBlockEst>>>(d_datosNivel0->areaYCosPhi, d_acumulador_1,
			d_deltaTVolumenesNivel0, numVolxNivel0, numVolyNivel0, CFL);
	delta_T = obtenerMinimoReduccion<double>(d_deltaTVolumenesNivel0, numVolxNivel0*numVolyNivel0);

	return delta_T;
}

double siguientePasoNivel0(double2 *d_datosVolumenesNivel0_1, double2 *d_datosVolumenesNivel0_2, tipoDatosSubmalla *d_datosNivel0,
		double2 *d_acumuladorNivel_1, double2 *d_acumuladorNivel_2, int numVolxNivel0, int numVolyNivel0, double *d_deltaTVolumenesNivel0,
		double borde_sup, double borde_inf, double borde_izq, double borde_der, double Hmin, int tam_spongeSup, int tam_spongeInf,
		int tam_spongeIzq, int tam_spongeDer, double sea_level, double *tiempo_act, double CFL, double delta_T, double mf0, double vmax,
		double epsilon_h, double hpos, double cvis, double L, double H, int64_t tam_datosVolDouble2Nivel0, dim3 blockGridVer1,
		dim3 blockGridVer2, dim3 blockGridHor1, dim3 blockGridHor2, dim3 threadBlockAri, dim3 blockGridEst, dim3 threadBlockEst)
{
	// PASO 1
	procesarAristasVerNivel0Paso1GPU<<<blockGridVer1, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->altoVolumenes, numVolxNivel0, numVolyNivel0, borde_izq, borde_der, delta_T,
		d_acumuladorNivel_1, CFL, epsilon_h, hpos, cvis, 1);
	procesarAristasVerNivel0Paso1GPU<<<blockGridVer2, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->altoVolumenes, numVolxNivel0, numVolyNivel0, borde_izq, borde_der, delta_T,
		d_acumuladorNivel_1, CFL, epsilon_h, hpos, cvis, 2);
	procesarAristasHorNivel0Paso1GPU<<<blockGridHor1, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->anchoVolumenes, numVolxNivel0, numVolyNivel0, borde_sup, borde_inf, delta_T,
		d_acumuladorNivel_1, CFL, epsilon_h, hpos, cvis, 1);
	procesarAristasHorNivel0Paso1GPU<<<blockGridHor2, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->anchoVolumenes, numVolxNivel0, numVolyNivel0, borde_sup, borde_inf, delta_T,
		d_acumuladorNivel_1, CFL, epsilon_h, hpos, cvis, 2);
	obtenerEstadosPaso1Nivel0GPU<<<blockGridEst, threadBlockEst>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->anchoVolumenes, d_datosNivel0->altoVolumenes, d_acumuladorNivel_1,
		numVolxNivel0, numVolyNivel0, delta_T, Hmin, tam_spongeSup, tam_spongeInf, tam_spongeIzq, tam_spongeDer, sea_level);

	hipMemset(d_acumuladorNivel_1, 0, tam_datosVolDouble2Nivel0);

	// PASO 2
	procesarAristasVerNivel0Paso2GPU<<<blockGridVer1, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->altoVolumenes, numVolxNivel0, numVolyNivel0, borde_izq, borde_der,
		delta_T, d_acumuladorNivel_1, d_acumuladorNivel_2, CFL, epsilon_h, hpos, cvis, 1);
	procesarAristasVerNivel0Paso2GPU<<<blockGridVer2, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->altoVolumenes, numVolxNivel0, numVolyNivel0, borde_izq, borde_der,
		delta_T, d_acumuladorNivel_1, d_acumuladorNivel_2, CFL, epsilon_h, hpos, cvis, 2);
	procesarAristasHorNivel0Paso2GPU<<<blockGridHor1, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->anchoVolumenes, numVolxNivel0, numVolyNivel0, borde_sup, borde_inf,
		delta_T, d_acumuladorNivel_1, d_acumuladorNivel_2, CFL, epsilon_h, hpos, cvis, 1);
	procesarAristasHorNivel0Paso2GPU<<<blockGridHor2, threadBlockAri>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->anchoVolumenes, numVolxNivel0, numVolyNivel0, borde_sup, borde_inf,
		delta_T, d_acumuladorNivel_1, d_acumuladorNivel_2, CFL, epsilon_h, hpos, cvis, 2);
	obtenerEstadoYDeltaTVolumenesNivel0GPU<<<blockGridEst, threadBlockEst>>>(d_datosVolumenesNivel0_1, d_datosVolumenesNivel0_2,
		d_datosNivel0->areaYCosPhi, d_datosNivel0->anchoVolumenes, d_datosNivel0->altoVolumenes, d_acumuladorNivel_1,
		d_acumuladorNivel_2, d_deltaTVolumenesNivel0, numVolxNivel0, numVolyNivel0, CFL, delta_T, mf0, vmax, hpos,
		epsilon_h, Hmin, tam_spongeSup, tam_spongeInf, tam_spongeIzq, tam_spongeDer, sea_level);

	*tiempo_act += delta_T;

	delta_T = obtenerMinimoReduccion<double>(d_deltaTVolumenesNivel0, numVolxNivel0*numVolyNivel0);

	return delta_T;
}

extern "C" int shallowWater(int numNiveles, int okada_flag, double LON_C, double LAT_C, double DEPTH_C, double FAULT_L,
			double FAULT_W, double STRIKE, double DIP, double RAKE, double SLIP, double2 *datosVolumenesNivel_1,
			double2 *datosVolumenesNivel_2, tipoDatosSubmalla datosNivel, int leer_fichero_puntos, int numPuntosGuardar,
			int *posicionesVolumenesGuardado, double *lonPuntos, double *latPuntos, int numVolxNivel0, int numVolyNivel0,
			int64_t numVolumenesNivel, double Hmin, char *nombre_bati, string prefijo, double borde_sup, double borde_inf,
			double borde_izq, double borde_der, int tam_spongeSup, int tam_spongeInf, int tam_spongeIzq, int tam_spongeDer,
			double tiempo_tot, double tiempoGuardarNetCDF, double tiempoGuardarSeries, double CFL, double mf0, double vmax,
			double epsilon_h, double hpos, double cvis, double L, double H, double Q, double T, double *tiempo)
{
	double2 *d_datosVolumenesNivel_1;
	double2 *d_datosVolumenesNivel_2;
	double2 *d_acumuladorNivel_1, *d_acumuladorNivel_2;
	tipoDatosSubmalla d_datosNivel;
	double *d_deltaTVolumenesNivel;
	double *d_eta1MaximaNivel;
	float *vec;
	double *p_eta;
	hipError_t err;
	int num = 0;
	double sea_level = SEA_LEVEL/H;
	dim3 blockGridVer1Nivel;
	dim3 blockGridVer2Nivel;
	dim3 blockGridHor1Nivel;
	dim3 blockGridHor2Nivel;
	dim3 blockGridEstNivel;
	dim3 threadBlockAri(NUM_HEBRASX_ARI, NUM_HEBRASY_ARI);
	dim3 threadBlockEst(NUM_HEBRASX_EST, NUM_HEBRASY_EST);
	dim3 blockGridPuntos(iDivUp(numPuntosGuardar, NUM_HEBRAS_PUNTOS), 1);
	dim3 threadBlockPuntos(NUM_HEBRAS_PUNTOS, 1);
	double lon_ini, lat_ini;
	double incx, incy;

	int *d_posicionesVolumenesGuardado;
	double2 *d_datosVolumenesGuardado_1, *d_datosVolumenesGuardado_2;
	float *etaPuntosGuardado;
	float *uPuntosGuardado;
	float *vPuntosGuardado;
	float *etaMinPuntosGuardado, *etaMaxPuntosGuardado;
	int num_ts = 0;

	int64_t tam_datosVolDoubleNivel;
	int64_t tam_datosVolDouble2Nivel;
	int64_t tam_datosDeltaTDouble;
	int64_t tam_datosAcumDouble2Nivel;
	int64_t tam_datosVolGuardadoDouble2 = ((int64_t) numPuntosGuardar)*sizeof(double2);
	double deltaTNivel;
	double tiempoActSubmalla;
	double sigTiempoGuardarNetCDF = 0.0;
	double sigTiempoGuardarSeries = 0.0;
	StopWatchInterface *timer = NULL;
	int i, iter;

	hipSetDevice(0);

	tiempoActSubmalla = 0.0;
	obtenerTamBloquesKernel(numVolxNivel0, numVolyNivel0, &blockGridVer1Nivel, &blockGridVer2Nivel,
		&blockGridHor1Nivel, &blockGridHor2Nivel, &blockGridEstNivel);

	tam_datosDeltaTDouble = 0;
	tam_datosAcumDouble2Nivel = 0;
	hipMalloc((void **)&(d_datosNivel.areaYCosPhi), numVolyNivel0*sizeof(double2));
	hipMalloc((void **)&(d_datosNivel.anchoVolumenes), (numVolyNivel0+1)*sizeof(double));
	hipMalloc((void **)&(d_datosNivel.altoVolumenes), numVolyNivel0*sizeof(double));
	tam_datosVolDoubleNivel = numVolumenesNivel*sizeof(double);
	tam_datosVolDouble2Nivel = numVolumenesNivel*sizeof(double2);
	tam_datosDeltaTDouble = max(tam_datosDeltaTDouble, tam_datosVolDoubleNivel);
	tam_datosAcumDouble2Nivel = max(tam_datosAcumDouble2Nivel, tam_datosVolDouble2Nivel);
	hipMalloc((void **)&d_datosVolumenesNivel_1, tam_datosVolDouble2Nivel);
	hipMalloc((void **)&d_datosVolumenesNivel_2, tam_datosVolDouble2Nivel);
	hipMalloc((void **)&d_eta1MaximaNivel, tam_datosVolDoubleNivel);
	hipMalloc((void **)&d_deltaTVolumenesNivel, tam_datosDeltaTDouble);
	hipMalloc((void **)&d_acumuladorNivel_1, tam_datosAcumDouble2Nivel);
	err = hipMalloc( (void **)&d_acumuladorNivel_2, tam_datosAcumDouble2Nivel);
	if (err == hipErrorOutOfMemory) {
		hipFree(d_datosNivel.areaYCosPhi);
		hipFree(d_datosNivel.anchoVolumenes);
		hipFree(d_datosNivel.altoVolumenes);
		hipFree(d_datosVolumenesNivel_1);
		hipFree(d_datosVolumenesNivel_2);
		hipFree(d_eta1MaximaNivel);
		hipFree(d_deltaTVolumenesNivel);
		hipFree(d_acumuladorNivel_1);
		return 1;
	}
	if (leer_fichero_puntos == 1) {
		hipMalloc((void **)&d_posicionesVolumenesGuardado, ((int64_t) numPuntosGuardar)*sizeof(int));
		hipMalloc((void **)&d_datosVolumenesGuardado_1, tam_datosVolGuardadoDouble2);
		err = hipMalloc((void **)&d_datosVolumenesGuardado_2, tam_datosVolGuardadoDouble2);
		if (err == hipErrorOutOfMemory) {
			liberarMemoria(numNiveles, d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, d_datosNivel,
				d_eta1MaximaNivel, d_deltaTVolumenesNivel, d_acumuladorNivel_1, d_acumuladorNivel_2,
				1, d_posicionesVolumenesGuardado, d_datosVolumenesGuardado_1, d_datosVolumenesGuardado_2);
			return 1;
		}
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(procesarAristasVerDeltaTInicialNivel0GPU), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(procesarAristasHorDeltaTInicialNivel0GPU), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(procesarAristasVerNivel0Paso1GPU), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(procesarAristasVerNivel0Paso2GPU), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(procesarAristasHorNivel0Paso1GPU), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(procesarAristasHorNivel0Paso2GPU), hipFuncCachePreferL1);

	hipMemcpy(d_datosNivel.areaYCosPhi, datosNivel.areaYCosPhi, numVolyNivel0*sizeof(double2), hipMemcpyHostToDevice);
	hipMemcpy(d_datosNivel.anchoVolumenes, datosNivel.anchoVolumenes, (numVolyNivel0+1)*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_datosNivel.altoVolumenes, datosNivel.altoVolumenes, numVolyNivel0*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_datosVolumenesNivel_1, datosVolumenesNivel_1, tam_datosVolDouble2Nivel, hipMemcpyHostToDevice);
	hipMemcpy(d_datosVolumenesNivel_2, datosVolumenesNivel_2, tam_datosVolDouble2Nivel, hipMemcpyHostToDevice);
	if (leer_fichero_puntos == 1) {
		hipMemcpy(d_posicionesVolumenesGuardado, posicionesVolumenesGuardado, ((int64_t) numPuntosGuardar)*sizeof(int), hipMemcpyHostToDevice);
	}

	if (okada_flag == OKADA_STANDARD) {
		lon_ini = datosNivel.longitud[0];
		lat_ini = datosNivel.latitud[0];
		incx = (datosNivel.longitud[numVolxNivel0-1] - datosNivel.longitud[0])/(numVolxNivel0-1);
		incy = (datosNivel.latitud[numVolyNivel0-1] - datosNivel.latitud[0])/(numVolyNivel0-1);
		fprintf(stdout, "Applying Okada\n");
		aplicarOkadaStandardGPU<<<blockGridEstNivel, threadBlockEst>>>(d_datosVolumenesNivel_1, numVolxNivel0, numVolyNivel0,
			lon_ini, incx, lat_ini, incy, LON_C, LAT_C, DEPTH_C, FAULT_L, FAULT_W, STRIKE, DIP, RAKE, SLIP, H);
	}
	hipDeviceSynchronize();

	inicializarEta1MaximaNivel0GPU<<<blockGridEstNivel, threadBlockEst>>>(d_datosVolumenesNivel_1,
		d_eta1MaximaNivel, numVolxNivel0, numVolyNivel0, epsilon_h);

	// INICIO NETCDF
	double mf0_ini = sqrt(mf0*pow(H,4.0/3.0)/(9.81*L));
	if (tiempoGuardarNetCDF >= 0.0) {
		vec = (float *) malloc(tam_datosDeltaTDouble);
		if (vec == NULL) {
			liberarMemoria(numNiveles, d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, d_datosNivel,
				d_eta1MaximaNivel, d_deltaTVolumenesNivel, d_acumuladorNivel_1, d_acumuladorNivel_2,
				leer_fichero_puntos, d_posicionesVolumenesGuardado, d_datosVolumenesGuardado_1,
				d_datosVolumenesGuardado_2);
			return 2;
		}
		for (i=0; i<numVolumenesNivel; i++)
			vec[i] = (float) ((datosVolumenesNivel_1[i].y + Hmin)*H);
		iter = crearFicherosNC(nombre_bati, okada_flag, (char *) prefijo.c_str(), numVolxNivel0, numVolyNivel0,
					datosNivel.longitud, datosNivel.latitud, tiempo_tot*T, CFL, epsilon_h*H, mf0_ini, vmax*Q/H,
					hpos*H, 1.0-cvis, borde_sup, borde_inf, borde_izq, borde_der, LON_C, LAT_C, DEPTH_C,
					FAULT_L, FAULT_W, STRIKE, DIP, RAKE, SLIP, vec);
		if (iter == 1) {
			liberarMemoria(numNiveles, d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, d_datosNivel,
				d_eta1MaximaNivel, d_deltaTVolumenesNivel, d_acumuladorNivel_1, d_acumuladorNivel_2,
				leer_fichero_puntos, d_posicionesVolumenesGuardado, d_datosVolumenesGuardado_1,
				d_datosVolumenesGuardado_2);
			free(vec);
			return 2;
		}
	}
	if (leer_fichero_puntos == 1) {
		etaPuntosGuardado = (float *) malloc(numPuntosGuardar*sizeof(float));
		uPuntosGuardado = (float *) malloc(numPuntosGuardar*sizeof(float));
		vPuntosGuardado = (float *) malloc(numPuntosGuardar*sizeof(float));
		etaMinPuntosGuardado = (float *) malloc(numPuntosGuardar*sizeof(float));
		etaMaxPuntosGuardado = (float *) malloc(numPuntosGuardar*sizeof(float));
		if (etaMaxPuntosGuardado == NULL) {
			if (etaPuntosGuardado != NULL)			free(etaPuntosGuardado);
			if (uPuntosGuardado != NULL)			free(uPuntosGuardado);
			if (vPuntosGuardado != NULL)			free(vPuntosGuardado);
			if (etaMinPuntosGuardado != NULL)		free(etaMinPuntosGuardado);
			if (tiempoGuardarNetCDF >= 0.0)
				free(vec);
			liberarMemoria(numNiveles, d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, d_datosNivel,
				d_eta1MaximaNivel, d_deltaTVolumenesNivel, d_acumuladorNivel_1, d_acumuladorNivel_2,
				leer_fichero_puntos, d_posicionesVolumenesGuardado, d_datosVolumenesGuardado_1,
				d_datosVolumenesGuardado_2);
			return 2;
		}
		initTimeSeriesNC(nombre_bati, (char *) prefijo.c_str(), numPuntosGuardar, lonPuntos, latPuntos,
			tiempo_tot*T, CFL, epsilon_h*H, mf0_ini, vmax*Q/H, hpos*H, 1.0-cvis, borde_sup, borde_inf,
			borde_izq, borde_der, LON_C, LAT_C, DEPTH_C, FAULT_L, FAULT_W, STRIKE, DIP, RAKE, SLIP, okada_flag);
		for (i=0; i<numPuntosGuardar; i++) {
			if (posicionesVolumenesGuardado[i] == -1) {
				etaPuntosGuardado[i] = -9999.0f;
				uPuntosGuardado[i] = -9999.0f;
				vPuntosGuardado[i] = -9999.0f;
				etaMinPuntosGuardado[i] = -9999.0f;
				etaMaxPuntosGuardado[i] = -9999.0f;
			}
			else {
				etaMinPuntosGuardado[i] = 1e30f;
				etaMaxPuntosGuardado[i] = -1e30f;
			}
		}
	}
	// FIN NETCDF

	hipMemset(d_acumuladorNivel_1, 0, tam_datosVolDouble2Nivel);
	hipMemset(d_acumuladorNivel_2, 0, tam_datosVolDouble2Nivel);

	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	deltaTNivel = obtenerDeltaTInicialNivel0(d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, &d_datosNivel,
					d_deltaTVolumenesNivel, d_acumuladorNivel_1, numVolxNivel0, numVolyNivel0, borde_izq,
					borde_der, borde_sup, borde_inf, CFL, epsilon_h, blockGridVer1Nivel, blockGridVer2Nivel,
					blockGridHor1Nivel, blockGridHor2Nivel, threadBlockAri, blockGridEstNivel, threadBlockEst);
	fprintf(stdout, "Initial deltaT = %e sec\n", deltaTNivel*T);

	hipMemset(d_acumuladorNivel_1, 0, tam_datosVolDouble2Nivel);

	iter = 1;
	while (tiempoActSubmalla < tiempo_tot) {
		// INICIO NETCDF
		if ((tiempoGuardarNetCDF >= 0.0) && (tiempoActSubmalla >= sigTiempoGuardarNetCDF)) {
			sigTiempoGuardarNetCDF += tiempoGuardarNetCDF;
			hipMemcpy(datosVolumenesNivel_1, d_datosVolumenesNivel_1, tam_datosVolDouble2Nivel, hipMemcpyDeviceToHost);
			hipMemcpy(datosVolumenesNivel_2, d_datosVolumenesNivel_2, tam_datosVolDouble2Nivel, hipMemcpyDeviceToHost);
			guardarNetCDFNivel0(datosVolumenesNivel_1, datosVolumenesNivel_2, vec, num, numVolxNivel0, numVolyNivel0,
				Hmin, tiempoActSubmalla, epsilon_h, H, Q, T);
			num++;
		}
		if ((tiempoGuardarSeries >= 0.0) && (tiempoActSubmalla >= sigTiempoGuardarSeries)) {
			sigTiempoGuardarSeries += tiempoGuardarSeries;
			escribirVolumenesGuardadoGPU<<<blockGridPuntos, threadBlockPuntos>>>(d_datosVolumenesNivel_1,
				d_datosVolumenesNivel_2, d_datosVolumenesGuardado_1, d_datosVolumenesGuardado_2,
				d_posicionesVolumenesGuardado, numPuntosGuardar, epsilon_h);
			hipMemcpy(datosVolumenesNivel_1, d_datosVolumenesGuardado_1, tam_datosVolGuardadoDouble2, hipMemcpyDeviceToHost);
			hipMemcpy(datosVolumenesNivel_2, d_datosVolumenesGuardado_2, tam_datosVolGuardadoDouble2, hipMemcpyDeviceToHost);
			guardarSerieTiemposNivel0(datosVolumenesNivel_1, datosVolumenesNivel_2, numPuntosGuardar, posicionesVolumenesGuardado,
				etaPuntosGuardado, uPuntosGuardado, vPuntosGuardado, etaMinPuntosGuardado, etaMaxPuntosGuardado,
				Hmin, num_ts, tiempoActSubmalla, H, Q, T);
			num_ts++;
		}
		// FIN NETCDF
		actualizarEta1MaximaNivel0GPU<<<blockGridEstNivel, threadBlockEst>>>(d_datosVolumenesNivel_1,
			d_eta1MaximaNivel, numVolxNivel0, numVolyNivel0, tiempoActSubmalla, epsilon_h);

		deltaTNivel = siguientePasoNivel0(d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, &d_datosNivel, d_acumuladorNivel_1,
						d_acumuladorNivel_2, numVolxNivel0, numVolyNivel0, d_deltaTVolumenesNivel, borde_sup, borde_inf,
						borde_izq, borde_der, Hmin, tam_spongeSup, tam_spongeInf, tam_spongeIzq, tam_spongeDer, sea_level,
						&tiempoActSubmalla, CFL, deltaTNivel, mf0, vmax, epsilon_h, hpos, cvis, L, H, tam_datosVolDouble2Nivel,
						blockGridVer1Nivel, blockGridVer2Nivel, blockGridHor1Nivel, blockGridHor2Nivel, threadBlockAri,
						blockGridEstNivel, threadBlockEst);

		hipMemset(d_acumuladorNivel_1, 0, tam_datosVolDouble2Nivel);
		hipMemset(d_acumuladorNivel_2, 0, tam_datosVolDouble2Nivel);

		fprintf(stdout, "Iteration %3d, deltaT = %e sec, ", iter, deltaTNivel*T);
		fprintf(stdout, "Time = %g sec\n", tiempoActSubmalla*T);
		iter++;
	}
	hipDeviceSynchronize();
	sdkStopTimer(&timer);
	*tiempo = sdkGetTimerValue(&timer)*0.001;

	// INICIO NETCDF
	if (tiempoGuardarNetCDF >= 0.0) {
		if (okada_flag == OKADA_STANDARD) {
			hipMemcpy(datosVolumenesNivel_1, d_datosVolumenesNivel_1, tam_datosVolDouble2Nivel, hipMemcpyDeviceToHost);
			for (i=0; i<numVolumenesNivel; i++)
				vec[i] = (float) ((datosVolumenesNivel_1[i].y + Hmin)*H);
			guardarBatimetriaModificadaNC(vec);
		}

		hipMemcpy(datosVolumenesNivel_1, d_eta1MaximaNivel, tam_datosVolDoubleNivel, hipMemcpyDeviceToHost);
		p_eta = (double *) datosVolumenesNivel_1;
		for (i=0; i<numVolumenesNivel; i++) {
			if (p_eta[i] < -1e20)
				vec[i] = -9999.0f;
			else
				vec[i] = (float) ((p_eta[i] - Hmin)*H);
		}
		cerrarFicheroNC(vec);
		free(vec);
	}
	if (leer_fichero_puntos == 1) {
		escribirVolumenesGuardadoGPU<<<blockGridPuntos, threadBlockPuntos>>>(d_datosVolumenesNivel_1,
			d_datosVolumenesNivel_2, d_datosVolumenesGuardado_1, d_datosVolumenesGuardado_2,
			d_posicionesVolumenesGuardado, numPuntosGuardar, epsilon_h);
		hipMemcpy(datosVolumenesNivel_1, d_datosVolumenesGuardado_1, tam_datosVolGuardadoDouble2, hipMemcpyDeviceToHost);
		obtenerBatimetriaParaSerieTiempos(datosVolumenesNivel_1, numPuntosGuardar, posicionesVolumenesGuardado,
			etaPuntosGuardado, Hmin, H);
		guardarBatimetriaModificadaTimeSeriesNC(etaPuntosGuardado);
		guardarAmplitudesTimeSeriesNC(etaMinPuntosGuardado, etaMaxPuntosGuardado);
		closeTimeSeriesNC();
		free(etaPuntosGuardado);
		free(uPuntosGuardado);
		free(vPuntosGuardado);
		free(etaMinPuntosGuardado);
		free(etaMaxPuntosGuardado);
	}
	// FIN NETCDF

	sdkDeleteTimer(&timer);
	liberarMemoria(numNiveles, d_datosVolumenesNivel_1, d_datosVolumenesNivel_2, d_datosNivel,
		d_eta1MaximaNivel, d_deltaTVolumenesNivel, d_acumuladorNivel_1, d_acumuladorNivel_2,
		leer_fichero_puntos, d_posicionesVolumenesGuardado, d_datosVolumenesGuardado_1,
		d_datosVolumenesGuardado_2);

	return 0;
}

