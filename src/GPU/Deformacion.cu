#include "hip/hip_runtime.h"
#ifndef _DEFORMACION_H_
#define _DEFORMACION_H_

#include <stdio.h>
#include "Constantes.hxx"
#include "Reduccion_kernel.cu"
#include "prtoxy.cu"
#include "DC3D.cu"

/******************/
/* Okada standard */
/******************/

__global__ void convertirAMenosValorAbsolutoGPU(double *d_in, float *d_out, int n)
{
	int pos = blockIdx.x*NUM_HEBRAS_PUNTOS + threadIdx.x;

	if (pos < n) {
		d_out[pos] = (float) (-fabs(d_in[pos]));
	}
}

__global__ void truncarDeformacionGPU(double *d_def, int num_volx, int num_voly, double crop_value)
{
	int pos, pos_x_hebra, pos_y_hebra;
	double U_Z;

	pos_x_hebra = blockIdx.x*NUM_HEBRASX_EST + threadIdx.x;
	pos_y_hebra = blockIdx.y*NUM_HEBRASY_EST + threadIdx.y;

	if ((pos_x_hebra < num_volx) && (pos_y_hebra < num_voly)) {
		pos = pos_y_hebra*num_volx + pos_x_hebra;
		U_Z = d_def[pos];
		if (fabs(U_Z) < crop_value)
			d_def[pos] = 0.0;
	}
}

__global__ void sumarDeformacionADatosGPU(double2 *d_datosVolumenes_1, double *d_def, double *d_eta1Inicial,
				int num_volx, int num_voly)
{
	int pos, pos_x_hebra, pos_y_hebra;
	double U_Z;

	pos_x_hebra = blockIdx.x*NUM_HEBRASX_EST + threadIdx.x;
	pos_y_hebra = blockIdx.y*NUM_HEBRASY_EST + threadIdx.y;

	if ((pos_x_hebra < num_volx) && (pos_y_hebra < num_voly)) {
		pos = pos_y_hebra*num_volx + pos_x_hebra;
		U_Z = d_def[pos];
		d_datosVolumenes_1[pos].y -= U_Z;
		d_eta1Inicial[pos] += U_Z;
	}
}

__global__ void aplicarOkadaStandardGPU(double2 *d_datosVolumenes_1, double *d_def, int num_volx, int num_voly, double lon_ini,
				double incx, double lat_ini, double incy, double LON_C_ent, double LAT_C_ent, double DEPTH_C_ent, double FAULT_L,
				double FAULT_W, double STRIKE, double DIP_ent, double RAKE, double SLIP, double H)
{
	double LON_P, LAT_P;
	double LON_C, LAT_C;
	double DEPTH_C, DIP;
	double S_RAKE;
	double C_RAKE;
	double S_STRIKE;
	double C_STRIKE;
	double Z;
	double AL1, AL2, AW1, AW2;
	double X_OKA, Y_OKA;
	double XP, YP;
	double RAD = M_PI/180.0;
	double alfa = 2.0/3.0;
	int i0 = 0;
	int IRET;
	double U_X, U_Y, U_Z, UXX, UYX, UZX, UXY, UYY, UZY, UXZ, UYZ, UZZ;
	double DISL1, DISL2, DISL3;
	int pos, pos_x_hebra, pos_y_hebra;

	pos_x_hebra = blockIdx.x*NUM_HEBRASX_EST + threadIdx.x;
	pos_y_hebra = blockIdx.y*NUM_HEBRASY_EST + threadIdx.y;

	if ((pos_x_hebra < num_volx) && (pos_y_hebra < num_voly)) {
		pos = pos_y_hebra*num_volx + pos_x_hebra;
		LON_C = LON_C_ent;
		LAT_C = LAT_C_ent;
		DEPTH_C = DEPTH_C_ent;
		DIP = DIP_ent;

		LON_P = lon_ini + pos_x_hebra*incx;
		LAT_P = lat_ini + pos_y_hebra*incy;

		S_RAKE = sin(RAD*RAKE);
		C_RAKE = cos(RAD*RAKE);

		S_STRIKE = sin(RAD*STRIKE);
		C_STRIKE = cos(RAD*STRIKE);

		DISL2 = SLIP*S_RAKE;
		DISL1 = SLIP*C_RAKE;
		DISL3 = 0.0;

		Z = 0.0;
		AL1 = -0.5*FAULT_L;
		AL2 = 0.5*FAULT_L;
		AW1 = -0.5*FAULT_W;
		AW2 = 0.5*FAULT_W;

		prtoxy_(&LAT_P, &LON_P, &LAT_C, &LON_C, &XP, &YP, &i0);
		X_OKA = XP*S_STRIKE + YP*C_STRIKE;
		Y_OKA = -XP*C_STRIKE + YP*S_STRIKE;
		dc3d_(&alfa, &X_OKA, &Y_OKA, &Z, &DEPTH_C, &DIP, &AL1, &AL2, &AW1, &AW2, &DISL1, &DISL2, &DISL3,
			&U_X, &U_Y, &U_Z, &UXX, &UYX, &UZX, &UXY, &UYY, &UZY, &UXZ, &UYZ, &UZZ, &IRET);

		U_Z /= H;
		d_def[pos] = U_Z;
	}
}

void aplicarOkada(double2 *d_datosVolumenes_1, double *d_eta1Inicial, int crop_flag, double crop_value,
		double *d_deltaTVolumenes, float *d_vec, int num_volx, int num_voly, double lon_ini, double incx,
		double lat_ini, double incy, double LON_C, double LAT_C, double DEPTH_C, double FAULT_L, double FAULT_W,
		double STRIKE, double DIP, double RAKE, double SLIP, dim3 blockGridEstNivel, dim3 threadBlockEst, double H)
{
	int num_volumenes = num_volx*num_voly;
	dim3 blockGridVec(iDivUp(num_volumenes, NUM_HEBRAS_PUNTOS), 1);
	dim3 threadBlockVec(NUM_HEBRAS_PUNTOS, 1);
	float def_max;
	double crop_value_final;

	aplicarOkadaStandardGPU<<<blockGridEstNivel, threadBlockEst>>>(d_datosVolumenes_1, d_deltaTVolumenes,
		num_volx, num_voly, lon_ini, incx, lat_ini, incy, LON_C, LAT_C, DEPTH_C, FAULT_L, FAULT_W,
		STRIKE, DIP, RAKE, SLIP, H);
	if (crop_flag == CROP_RELATIVE) {
		convertirAMenosValorAbsolutoGPU<<<blockGridVec, threadBlockVec>>>(d_deltaTVolumenes, d_vec, num_volumenes);
		def_max = -obtenerMinimoReduccion<float>(d_vec, num_volumenes);
		crop_value_final = crop_value*((double) def_max);
		truncarDeformacionGPU<<<blockGridEstNivel, threadBlockEst>>>(d_deltaTVolumenes, num_volx, num_voly, crop_value_final);
	}
	else if (crop_flag == CROP_ABSOLUTE) {
		truncarDeformacionGPU<<<blockGridEstNivel, threadBlockEst>>>(d_deltaTVolumenes, num_volx, num_voly, crop_value);
	}
	sumarDeformacionADatosGPU<<<blockGridEstNivel, threadBlockEst>>>(d_datosVolumenes_1, d_deltaTVolumenes,
		d_eta1Inicial, num_volx, num_voly);
}

#endif
