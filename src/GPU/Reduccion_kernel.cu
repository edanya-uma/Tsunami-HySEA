#include "hip/hip_runtime.h"
#ifndef _REDUCCION_KERNEL_H_
#define _REDUCCION_KERNEL_H_

#include "sharedmem.cuh"

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)
*/
template <class T, unsigned int blockSize>
__global__ void reduce6_min(T *g_idata, T *g_odata, unsigned int n)
{
    SharedMemory<T> smem;
    T *sdata = smem.getPointer();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridSize).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
	T thMin, val;

	if (i + blockSize >= n)
		thMin = g_idata[i];
	else
		thMin = fminf(g_idata[i], g_idata[i + blockSize]);
	i += gridSize;
	while (i < n) {
		if (i + blockSize >= n)
			val = g_idata[i];
		else
			val = fminf(g_idata[i], g_idata[i + blockSize]);
		thMin = fminf(thMin, val);
		i += gridSize;
	}
	sdata[tid] = thMin;

	__syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) {
		if (tid < 256) { sdata[tid] = fminf(sdata[tid], sdata[tid + 256]); }
		__syncthreads();
	}
    if (blockSize >= 256) {
		if (tid < 128) { sdata[tid] = fminf(sdata[tid], sdata[tid + 128]); }
		__syncthreads();
	}
    if (blockSize >= 128) {
		if (tid <  64) { sdata[tid] = fminf(sdata[tid], sdata[tid +  64]); }
		__syncthreads();
	}

	if (tid < 32) {
	        // now that we are using warp-synchronous programming (below)
	        // we need to declare our shared memory volatile so that the compiler
	        // doesn't reorder stores to it and induce incorrect behavior.
	        volatile T* smem = sdata;
	        if (blockSize >=  64) smem[tid] = fminf(smem[tid], smem[tid + 32]);
	        if (blockSize >=  32) smem[tid] = fminf(smem[tid], smem[tid + 16]);
	        if (blockSize >=  16) smem[tid] = fminf(smem[tid], smem[tid +  8]);
	        if (blockSize >=   8) smem[tid] = fminf(smem[tid], smem[tid +  4]);
	        if (blockSize >=   4) smem[tid] = fminf(smem[tid], smem[tid +  2]);
	        if (blockSize >=   2) smem[tid] = fminf(smem[tid], smem[tid +  1]);
	}

	// write result for this block to global mem 
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void reduce_min(int blocks, int threads, T *d_idata, T *d_odata, int size)
{
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);
	int smemSize = threads * sizeof(T);

	switch (threads) {
	case 512:
		reduce6_min<T, 512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case 256:
		reduce6_min<T, 256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case 128:
		reduce6_min<T, 128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case 64:
		reduce6_min<T,  64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case 32:
		reduce6_min<T,  32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case 16:
		reduce6_min<T,  16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case  8:
		reduce6_min<T,   8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case  4:
		reduce6_min<T,   4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case  2:
		reduce6_min<T,   2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	case  1:
		reduce6_min<T,   1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size); break;
	}
}

template
void reduce_min<float>(int blocks, int threads, float *d_idata, float *d_odata, int size);

template
void reduce_min<double>(int blocks, int threads, double *d_idata, double *d_odata, int size);

void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
/*	if (n == 1) 
		threads = 1;
	else*/
		threads = (n < maxThreads*2) ? n / 2 : maxThreads;
	blocks = n / (threads * 2);
	blocks = min(maxBlocks, blocks);
}

template <class T>
T obtenerMinimoReduccion(T *d_data, int size)
{
	int maxBlocks = 64, maxThreads = 128;
	int numBlocks, numThreads;
	int i;
	T h_data[4096];
	T minimo;

	if (size > 4096) {
		getNumBlocksAndThreads(size, maxBlocks, maxThreads, numBlocks, numThreads);
		reduce_min<T>(numBlocks, numThreads, d_data, d_data, size);

		int s = numBlocks;
		while (s > 4096) {
			getNumBlocksAndThreads(s, maxBlocks, maxThreads, numBlocks, numThreads);
			reduce_min<T>(numBlocks, numThreads, d_data, d_data, s);
			s = s / (numThreads*2);
		}

		hipMemcpy(h_data, d_data, numBlocks*sizeof(T), hipMemcpyDeviceToHost);
		minimo = (T) 1e30;
		for (i=0; i<numBlocks; i++) {
			if (h_data[i] < minimo)
				minimo = h_data[i];
		}
	}
	else {
		hipMemcpy(h_data, d_data, size*sizeof(T), hipMemcpyDeviceToHost);
		minimo = (T) 1e30;
		for (i=0; i<size; i++) {
			if (h_data[i] < minimo)
				minimo = h_data[i];
		}
	}

	return minimo;
}

template <class T>
T obtenerMinimoReduccionNoMod(T *d_idata, T *d_odata, int size)
{
	int maxBlocks = 64, maxThreads = 128;
	int numBlocks, numThreads;
	int i;
	T h_data[4096];
	T minimo;

	if (size > 4096) {
		getNumBlocksAndThreads(size, maxBlocks, maxThreads, numBlocks, numThreads);
		reduce_min<T>(numBlocks, numThreads, d_idata, d_odata, size);

		int s = numBlocks;
		while (s > 4096) {
			getNumBlocksAndThreads(s, maxBlocks, maxThreads, numBlocks, numThreads);
			reduce_min<T>(numBlocks, numThreads, d_idata, d_odata, s);
			s = s / (numThreads*2);
		}

		hipMemcpy(h_data, d_odata, numBlocks*sizeof(T), hipMemcpyDeviceToHost);
		minimo = (T) 1e30;
		for (i=0; i<numBlocks; i++) {
			if (h_data[i] < minimo)
				minimo = h_data[i];
		}
	}
	else {
		hipMemcpy(h_data, d_odata, size*sizeof(T), hipMemcpyDeviceToHost);
		minimo = (T) 1e30;
		for (i=0; i<size; i++) {
			if (h_data[i] < minimo)
				minimo = h_data[i];
		}
	}

	return minimo;
}

#endif
